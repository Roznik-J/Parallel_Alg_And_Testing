#include <tcMatrixMultiply.hpp>
#include <iostream>

int Kernel::Err::GetError(hipblasStatus_t error)
{
    switch (error)
    {
        case HIPBLAS_STATUS_SUCCESS:
            return 0;

        case HIPBLAS_STATUS_NOT_INITIALIZED:
            return 1;

        case HIPBLAS_STATUS_ALLOC_FAILED:
            return 2;

        case HIPBLAS_STATUS_INVALID_VALUE:
            return 3;

        case HIPBLAS_STATUS_ARCH_MISMATCH:
            return 4;

        case HIPBLAS_STATUS_MAPPING_ERROR:
            return 5;

        case HIPBLAS_STATUS_EXECUTION_FAILED:
            return 6;

        case HIPBLAS_STATUS_INTERNAL_ERROR:
            return 7;
    }
    return -1;
}

void Kernel::Err::PrintError(int anErr)
{
    switch (anErr)
    {
        case 0:
            //std::cout << "HIPBLAS_STATUS_SUCCESS" << std::endl;
            break;

        case 1:
            std::cout << "HIPBLAS_STATUS_NOT_INITIALIZED" << std::endl;
            break;

        case 2:
            std::cout << "HIPBLAS_STATUS_ALLOC_FAILED" << std::endl;
            break;

        case 3:
            std::cout << "HIPBLAS_STATUS_INVALID_VALUE" << std::endl;
            break;

        case 4:
            std::cout << "HIPBLAS_STATUS_ARCH_MISMATCH" << std::endl;
            break;

        case 5:
            std::cout << "HIPBLAS_STATUS_MAPPING_ERROR" << std::endl;
            break;

        case 6:
            std::cout << "HIPBLAS_STATUS_EXECUTION_FAILED" << std::endl;
            break;

        case 7:
            std::cout << "HIPBLAS_STATUS_INTERNAL_ERROR" << std::endl;
            break;
        default:
            std::cout << "<unknown>" << std::endl;
    }
}

int Kernel::Matrix::Multiply(hipblasHandle_t handle, hipblasOperation_t transa, hipblasOperation_t transb, 
int m, int n, int k, const float *alpha, const float *A, int lda, const float *B, int ldb, const float *beta, float *C, int ldc)
{
    auto leResult = hipblasSgemm(handle, transa, transb, m, n, k, alpha, A, lda, B, ldb, beta, C, ldc);
    int errorenum = Kernel::Err::GetError(leResult);
    return errorenum;
}
